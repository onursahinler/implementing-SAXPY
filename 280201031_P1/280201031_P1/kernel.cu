﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void saxpy_kernel(int n, float a, float* x, float* y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main() {
    int N;
    float A;

    // Getting input from user for the "N" and "A"
    printf("Please enter the size of the arrays(N): ");
    scanf("%d", &N);
    printf("Plese enter the scalar value(A): ");
    scanf("%f", &A);

    int size = N * sizeof(float);

    // Allocating memory for arrays on CPU
    float* h_x = (float*)malloc(size);
    float* h_y = (float*)malloc(size);

    // Initializing arrays with random values
    for (int i = 0; i < N; ++i) {
        h_x[i] = (float)rand() / RAND_MAX;
        h_y[i] = (float)rand() / RAND_MAX;
    }

    // Allocating memory for arrays on GPU
    float* d_x;
    float* d_y;

    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);

    // Copying data from CPU(Host) to GPU(Device)
    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    // Getting GPU device properties with using cudaGetDeviceProperties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("GPU Device Name: %s\n", prop.name);
    printf("Maximum Threads Per Block: %d\n", prop.maxThreadsPerBlock);

    // Running SAXPY kernel function with different configurations assigned by myself
    int blockDimensions[] = {128, 256, 512};
    int numberofConfigs = sizeof(blockDimensions) / sizeof(int);

    for (int i = 0; i < numberofConfigs; ++i) {
        int blockDimension = blockDimensions[i];
        int threadNumbers = (N + (blockDimension - 1)) / blockDimension;

        saxpy_kernel <<<threadNumbers, blockDimension >>>(N, A, d_x, d_y);

        // Copying the result back from GPU(Device) to CPU(Host)
        hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
        
        printf("(Block Dimension: %d, Thread Numbers per each block: %d) ", blockDimension, threadNumbers);
        printf("Result: ");
        for (int j = 0; j < N; ++j) {
            printf("%.6f | ", h_y[j]);
        }
        printf("\n");
    }

    // Freeing allocated memory
    free(h_x);
    free(h_y);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}
